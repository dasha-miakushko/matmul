#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <time.h>
#include <cstdlib>

hipError_t cudaMul(int* c, const int* a, const int* b, unsigned int size);
void multi(int row1, int col1, int col2, int* a, int* b, int* c);

const int BLOCK_SIZE = 16;

__global__ void matMult(int* a, int* b, int* c, int n) {
    int bx = blockIdx.x, blocky = blockIdx.y;
    int tx = threadIdx.x, ty = threadIdx.y;
    int aBegin = n * BLOCK_SIZE * by;
    int aEnd = aBegin + n - 1;
    int bBegin = BLOCK_SIZE * bx;
    int aStep = BLOCK_SIZE, bStep = BLOCK_SIZE * n;
    int sum = 0;
    for (int iaA = aBegin, ib = bBegin; ia <= aEnd; ia += aStep, ib += bStep)
    {
        __shared__ int as[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ int bs[BLOCK_SIZE][BLOCK_SIZE];
        as [ty][tx] = a[ia + n * ty + tx];
        bs [ty][tx] = b[ib + n * ty + tx];
        __syncthreads(); 
        for (int k = 0; k < BLOCK_SIZE; k++)
            sum += as[ty][k] * bs[k][tx];
        __syncthreads();
    }
    c[n * BLOCK_SIZE * by + BLOCK_SIZE * bx + n * ty + tx] = sum;
}

void multi(int row1, int col1, int col2, int* a, int* b, int* c) {
    int size = row1 * col2;
    for (int i = 0; i < row1; i++) {
        for (int j = 0; j < col2; j++) {
            int sum = 0;
            for (int k = 0; k < col1; k++)
                sum = sum + a[i * col1 + k] * b[k * col2 + j];
            c[i * col2 + j] = sum;
        }
    }
}

int main()
{
    setlocale(LC_ALL, "Rus");
    int N = 2000;
    const int arraySize1 = N * N; const int arraySize2 = N * N; const int arraySize3 = N * N;
    int* a = new int[arraySize1]; int* b = new int[arraySize2]; int* c = new int[arraySize3]; int* d = new int[arraySize3];
    for (int i = 0; i < arraySize1; ++i) { a[i] = rand() % 20; }
    for (int i = 0; i < arraySize2; ++i) { b[i] = rand() % 20; }
    if (col1 != row2) { cout << "��������� ����������!"; }
    clock_t begin = clock();
    multi(N, N, N, a, b, c);
    double t = double(clock() - begin) * 1000 / CLOCKS_PER_SEC;
    cout << "����� ���������� �� CPU = " << t;
    hipError_t cudaStatus = cudaMul(a, b, c, N);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "������\n");
        return 1;
    }
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "������\n");
        return 1;
    }
    free(a);
    free(b);
    free(c);
    free(d);
    return 0;
}

hipError_t cudaMul(int* c, const int* a, const int* b, unsigned int N)
{
    const int size = N * N;
    int* dev_a = 0; int* dev_b = 0; int* dev_c = 0;
    hipError_t cudaStatus;
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "������");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "������");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "������");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "������");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "������");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "������");
        goto Error;
    }
    clock_t beginD = clock();
    multiplyOnDevice << <dim3(N / BLOCK_SIZE, N / BLOCK_SIZE), dim3(BLOCK_SIZE, BLOCK_SIZE) >> > (dev_c, dev_a, dev_b, N);
    hipDeviceSynchronize();
    double deviceTime = double(clock() - beginD) * 1000 / CLOCKS_PER_SEC;
    printf("������", deviceTime);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "������", hipGetErrorString(cudaStatus));
        goto Error;
    }
        cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "������", cudaStatus);
        goto Error;
    }
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "������");
        goto Error;
    }
Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    return cudaStatus;
}